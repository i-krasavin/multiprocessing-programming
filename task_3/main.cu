#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

using namespace std;

double r(double value, int k)
{
    return (double)round((pow(2, k) * value)) / pow(2, k);
}

// define kernel
__global__ void customInt(double *x, double *y, double *result)
{
    int index = threadIdx.x;
    result[index] = (index + 1) * pow((y[index] - sin(x[index])), 2);
}

void runRungeKutta(int thereadCount, double *listX, double *listY, double h)
{
    double **fPointer = new double *[3];
    for (int i = 0; i < 3; i++)
    {
        fPointer[i] = new double[thereadCount];
    }

    double *listY1 = new double[thereadCount];
    listY1 = listY;

    while (r(listX[0], 2) < 1.0)
    {
        double *f0Pointer;
        hipMalloc((void **)&f0Pointer, sizeof(double) * thereadCount);
        customInt<<<1, thereadCount>>>(listX, listY, f0Pointer);
        hipMemcpy(&fPointer[0], f0Pointer, sizeof(int) * thereadCount, hipMemcpyDeviceToHost);

        double *f1Pointer;
        hipMalloc((void **)&f1Pointer, sizeof(double) * thereadCount);
        customInt<<<1, thereadCount>>>(listX, listY, f1Pointer);
        hipMemcpy(&fPointer[1], f1Pointer, sizeof(int) * thereadCount, hipMemcpyDeviceToHost);

        double *f2Pointer;
        hipMalloc((void **)&f2Pointer, sizeof(double) * thereadCount);
        customInt<<<1, thereadCount>>>(listX, listY, f2Pointer);
        hipMemcpy(&fPointer[2], f2Pointer, sizeof(double) * thereadCount, hipMemcpyDeviceToHost);

        for (int j = 0; j < thereadCount; j++)
        {
            listY[j] = listY1[j] + (fPointer[0][j] + 2.0 * fPointer[1][j] + fPointer[2][j]) / 6.0;
        }

        // increment
        listX[0] += h;
    }

    // output
    for (int e = 0; e < thereadCount; e++) {
        cout << listY[e] << " ";
    }
}

int main()
{
    int thereadsCount = 1000;
    double *listX = new double[thereadsCount];
    double *listY = new double[thereadsCount];
    double h = 0.05;

    for (int i = 0; i < thereadsCount; i++)
    {
        listX[i] = rand() * 500;
        listY[i] = rand() * 500;
    }

    runRungeKutta(thereadsCount, listX, listY, h);

    return 0;
}